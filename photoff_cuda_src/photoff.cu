#include "hip/hip_runtime.h"
#include "photoff.h"
#include <stdio.h>

__global__ void cropKernel(const uchar4* src,
                           uchar4* dst,
                           uint32_t src_width,
                           uint32_t src_height,
                           uint32_t dst_width,
                           uint32_t dst_height,
                           int crop_x,
                           int crop_y) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_width || y >= dst_height) return;

    int src_x = crop_x + x;
    int src_y = crop_y + y;

    if (src_x < src_width && src_y < src_height) {
        dst[y * dst_width + x] = src[src_y * src_width + src_x];
    } else {
        dst[y * dst_width + x] = make_uchar4(0, 0, 0, 0);
    }
}

__global__ void chromaKeyKernel(uchar4* buffer,
                                const uchar4* key_buffer,
                                uint32_t buffer_width,
                                uint32_t buffer_height,
                                uint32_t key_width,
                                uint32_t key_height,
                                int channel,
                                unsigned char threshold,
                                bool invert,
                                bool zero_all_channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= buffer_width || y >= buffer_height) return;
    
    int buffer_idx = y * buffer_width + x;
    
    if (x < key_width && y < key_height) {
        int key_idx = y * key_width + x;
        uchar4 keyPixel = key_buffer[key_idx];
        
        unsigned char channelValue;
        switch(channel) {
            case 0: channelValue = keyPixel.x; break; // R
            case 1: channelValue = keyPixel.y; break; // G
            case 2: channelValue = keyPixel.z; break; // B
            case 3: channelValue = keyPixel.w; break; // A
            default: channelValue = keyPixel.y; break; // Default to G
        }
        
        bool makeTransparent = invert ? 
                              (channelValue <= threshold) : 
                              (channelValue > threshold);
        
        if (makeTransparent) {
            if (zero_all_channels) {
                buffer[buffer_idx] = make_uchar4(0, 0, 0, 0); // Poner todo el píxel a 0
            } else {
                buffer[buffer_idx].w = 0; // Solo modificar alfa, manteniendo RGB intacto
            }
        }
    }
}


__global__ void grayscaleKernel(uchar4* buffer,
                               uint32_t width,
                               uint32_t height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 pixel = buffer[idx];
    
    if (pixel.w == 0) return;
    
    unsigned char gray = (unsigned char)(
        0.299f * pixel.x + 
        0.587f * pixel.y + 
        0.114f * pixel.z
    );
    
    buffer[idx].x = gray;
    buffer[idx].y = gray;
    buffer[idx].z = gray;
}

__device__ float gaussianWeight(float distance, float sigma) {
    return expf(-(distance * distance) / (2.0f * sigma * sigma));
}

__global__ void gaussianBlurKernel(const uchar4* src,
                                   uchar4* dst,
                                   uint32_t width,
                                   uint32_t height,
                                   float radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    float sigma = radius / 2.0f;
    
    int kernelSize = ceilf(radius * 3.0f);
    kernelSize = max(1, min(kernelSize, 25));
    
    float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f, sumA = 0.0f;
    float totalWeight = 0.0f;
    
    for (int ky = -kernelSize; ky <= kernelSize; ky++) {
        for (int kx = -kernelSize; kx <= kernelSize; kx++) {
            int sampleX = min(width - 1, max(0, x + kx));
            int sampleY = min(height - 1, max(0, y + ky));
            
            float distance = sqrtf((float)(kx * kx + ky * ky));
            
            if (distance > kernelSize) continue;
            
            float weight = gaussianWeight(distance, sigma);
            
            uchar4 sample = src[sampleY * width + sampleX];
            
            float alpha = sample.w / 255.0f;
            
            sumR += sample.x * weight * alpha;
            sumG += sample.y * weight * alpha;
            sumB += sample.z * weight * alpha;
            sumA += sample.w * weight;
            
            totalWeight += weight;
        }
    }
    
    if (totalWeight > 0.0f) {
        float alpha = sumA / (totalWeight * 255.0f);
        
        if (alpha > 0.0f) {
            dst[y * width + x].x = (unsigned char)(sumR / (totalWeight * alpha));
            dst[y * width + x].y = (unsigned char)(sumG / (totalWeight * alpha));
            dst[y * width + x].z = (unsigned char)(sumB / (totalWeight * alpha));
            dst[y * width + x].w = (unsigned char)(sumA / totalWeight);
        } else {
            dst[y * width + x] = make_uchar4(0, 0, 0, 0);
        }
    } else {
        dst[y * width + x] = make_uchar4(0, 0, 0, 0);
    }
}

__global__ void copyBufferKernel(uchar4* dst,
                                 const uchar4* src,
                                 uint32_t width,
                                 uint32_t height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    dst[idx] = src[idx];
}

__device__ float calculateShadowWeight(int x,
                                       int y,
                                       const uchar4* buffer,
                                       uint32_t width,
                                       uint32_t height,
                                       float radius,
                                       bool isInner) {
    float minDistance = radius;
    int r2 = radius * radius;
    
    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {
            if (dx*dx + dy*dy > r2) continue;
            
            int nx = x + dx;
            int ny = y + dy;
            
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                bool hasAlpha = buffer[ny * width + nx].w > 0;
                if (hasAlpha != isInner) {
                    float distance = sqrtf(dx*dx + dy*dy);
                    minDistance = min(minDistance, distance);
                }
            }
        }
    }
    
    float weight = 1.0f - (minDistance / radius);
    return max(0.0f, min(1.0f, weight));
}

__global__ void shadowKernel(const uchar4* src,
                             uchar4* dst,
                             uint32_t width,
                             uint32_t height,
                             float radius,
                             float intensity,
                             uchar4 shadow_color,
                             bool isInner) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 srcPixel = src[idx];
    
    if ((isInner && srcPixel.w == 0) || (!isInner && srcPixel.w > 0)) {
        dst[idx] = srcPixel;
        return;
    }
    
    float shadowWeight = calculateShadowWeight(x, y, src, width, height, radius, isInner);
    shadowWeight *= intensity;
    
    if (isInner) {
        float invWeight = 1.0f - shadowWeight;
        dst[idx].x = (unsigned char)(srcPixel.x * invWeight + shadow_color.x * shadowWeight);
        dst[idx].y = (unsigned char)(srcPixel.y * invWeight + shadow_color.y * shadowWeight);
        dst[idx].z = (unsigned char)(srcPixel.z * invWeight + shadow_color.z * shadowWeight);
        dst[idx].w = srcPixel.w;
    } else {
        if (shadowWeight > 0.0f) {
            float finalAlpha = shadow_color.w / 255.0f * shadowWeight;
            dst[idx].x = shadow_color.x;
            dst[idx].y = shadow_color.y;
            dst[idx].z = shadow_color.z;
            dst[idx].w = (unsigned char)(finalAlpha * 255.0f);
        } else {
            dst[idx] = make_uchar4(0, 0, 0, 0);
        }
    }
}

__device__ float bicubicWeight(float x, float a = -0.5f) {
    x = fabsf(x);
    if (x <= 1.0f) {
        return ((a + 2.0f) * x * x * x) - ((a + 3.0f) * x * x) + 1.0f;
    } else if (x < 2.0f) {
        return (a * x * x * x) - (5.0f * a * x * x) + (8.0f * a * x) - (4.0f * a);
    }
    return 0.0f;
}

__global__ void resizeBicubicKernel(uchar4* dst,
                                    const uchar4* src,
                                    uint32_t dst_width,
                                    uint32_t dst_height,
                                    uint32_t src_width,
                                    uint32_t src_height) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)(src_width) / dst_width;
    float scale_y = (float)(src_height) / dst_height;
    
    float src_x = dst_x * scale_x;
    float src_y = dst_y * scale_y;

    int x0 = floorf(src_x - 1.0f);
    int y0 = floorf(src_y - 1.0f);
    
    float4 result = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float totalWeight = 0.0f;

    #pragma unroll
    for (int dy = 0; dy < 4; dy++) {
        int sy = y0 + dy;
        float wy = bicubicWeight(src_y - sy);
        
        #pragma unroll
        for (int dx = 0; dx < 4; dx++) {
            int sx = x0 + dx;
            
            if (sx >= 0 && sx < src_width && sy >= 0 && sy < src_height) {
                float wx = bicubicWeight(src_x - sx);
                float weight = wx * wy;
                
                uchar4 pixel = src[sy * src_width + sx];
                result.x += weight * pixel.x;
                result.y += weight * pixel.y;
                result.z += weight * pixel.z;
                result.w += weight * pixel.w;
                totalWeight += weight;
            }
        }
    }

    if (totalWeight > 0.0f) {
        result.x = fmaxf(0.0f, fminf(255.0f, result.x / totalWeight));
        result.y = fmaxf(0.0f, fminf(255.0f, result.y / totalWeight));
        result.z = fmaxf(0.0f, fminf(255.0f, result.z / totalWeight));
        result.w = fmaxf(0.0f, fminf(255.0f, result.w / totalWeight));
    }

    dst[dst_y * dst_width + dst_x] = make_uchar4(
        __float2int_rn(result.x),
        __float2int_rn(result.y),
        __float2int_rn(result.z),
        __float2int_rn(result.w)
    );
}

__global__ void resizeBilinearKernel(uchar4* dst,
                                     const uchar4* src,
                                     uint32_t dst_width,
                                     uint32_t dst_height,
                                     uint32_t src_width,
                                     uint32_t src_height) {

    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)(src_width - 1) / dst_width;
    float scale_y = (float)(src_height - 1) / dst_height;
    
    float src_x = dst_x * scale_x;
    float src_y = dst_y * scale_y;
    
    int x1 = (int)src_x;
    int y1 = (int)src_y;
    int x2 = min(x1 + 1, (int)src_width - 1);
    int y2 = min(y1 + 1, (int)src_height - 1);
    
    float wx2 = src_x - x1;
    float wy2 = src_y - y1;
    float wx1 = 1.0f - wx2;
    float wy1 = 1.0f - wy2;
    
    uchar4 p11 = src[y1 * src_width + x1];
    uchar4 p21 = src[y1 * src_width + x2];
    uchar4 p12 = src[y2 * src_width + x1];
    uchar4 p22 = src[y2 * src_width + x2];
    
    int dst_idx = dst_y * dst_width + dst_x;
    dst[dst_idx].x = (unsigned char)(
        p11.x * wx1 * wy1 +
        p21.x * wx2 * wy1 +
        p12.x * wx1 * wy2 +
        p22.x * wx2 * wy2);
    
    dst[dst_idx].y = (unsigned char)(
        p11.y * wx1 * wy1 +
        p21.y * wx2 * wy1 +
        p12.y * wx1 * wy2 +
        p22.y * wx2 * wy2);
    
    dst[dst_idx].z = (unsigned char)(
        p11.z * wx1 * wy1 +
        p21.z * wx2 * wy1 +
        p12.z * wx1 * wy2 +
        p22.z * wx2 * wy2);
    
    dst[dst_idx].w = (unsigned char)(
        p11.w * wx1 * wy1 +
        p21.w * wx2 * wy1 +
        p12.w * wx1 * wy2 +
        p22.w * wx2 * wy2);
}

__global__ void resizeNearestKernel(uchar4* dst,
                                    const uchar4* src,
                                    uint32_t dst_width,
                                    uint32_t dst_height,
                                    uint32_t src_width,
                                    uint32_t src_height) {

    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)src_width / dst_width;
    float scale_y = (float)src_height / dst_height;

    int src_x = (int)(dst_x * scale_x);
    int src_y = (int)(dst_y * scale_y);
    
    dst[dst_y * dst_width + dst_x] = src[src_y * src_width + src_x];
}


__global__ void fillColorKernel(uchar4* buffer,
                                uchar4 color, 
                                uint32_t width,
                                uint32_t height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        buffer[idx] = color;
    }
}


__global__ void blendKernel(uchar4* dst,
                            const uchar4* src,
                            uint32_t dst_width,
                            uint32_t dst_height,
                            uint32_t src_width,
                            uint32_t src_height,
                            int32_t pos_x,
                            int32_t pos_y) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_width || y >= dst_height) return;

    int src_x = x - pos_x;
    int src_y = y - pos_y;

    if (src_x >= 0 && src_x < src_width && 
        src_y >= 0 && src_y < src_height) {
        
        int dst_idx = y * dst_width + x;
        int src_idx = src_y * src_width + src_x;
        
        uchar4 src_pixel = src[src_idx];
        uchar4 dst_pixel = dst[dst_idx];
        
        float srcA = src_pixel.w / 255.0f;
        float dstA = dst_pixel.w / 255.0f;

        // Case 1: pixel totally opaque
        if (src_pixel.w == 255) {
            dst[dst_idx] = src_pixel;
        } 
        // Case 2: pixel totally transparent
        else if (src_pixel.w == 0) {
            // Do nothing
        } 
        // Case 3: pixel semi-transparent
        else {
            float outA = srcA + dstA * (1.0f - srcA);

            // Avoid division by zero
            if (outA > 0.0f) {
                float outR = (src_pixel.x * srcA + dst_pixel.x * dstA * (1.0f - srcA)) / outA;
                float outG = (src_pixel.y * srcA + dst_pixel.y * dstA * (1.0f - srcA)) / outA;
                float outB = (src_pixel.z * srcA + dst_pixel.z * dstA * (1.0f - srcA)) / outA;

                dst[dst_idx].x = static_cast<unsigned char>(outR);
                dst[dst_idx].y = static_cast<unsigned char>(outG);
                dst[dst_idx].z = static_cast<unsigned char>(outB);
                dst[dst_idx].w = static_cast<unsigned char>(outA * 255.0f);
            } else {
                // Do nothing
            }
        }
    }
}

__global__ void cornerRadiusKernel(uchar4* buffer,
                                   uint32_t width,
                                   uint32_t height,
                                   uint32_t radius) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    
    if (x < radius && y < radius) {
        int dx = radius - 1 - x;
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y < radius) {
        int dx = x - (width - radius);
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x < radius && y >= height - radius) {
        int dx = radius - 1 - x;
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y >= height - radius) {
        int dx = x - (width - radius);
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }
}

__global__ void strokeKernel(const uchar4* src,
                             uchar4* dst,
                             uint32_t width,
                             uint32_t height,
                             int stroke_width,
                             uchar4 stroke_color) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 pixel = src[idx];

    if (pixel.w != 0) {
        dst[idx] = pixel;
        return;
    }
    
    int r2 = stroke_width * stroke_width;
    for (int dy = -stroke_width; dy <= stroke_width; dy++) {
        for (int dx = -stroke_width; dx <= stroke_width; dx++) {
            if (dx*dx + dy*dy > r2) continue;
            
            int nx = x + dx;
            int ny = y + dy;
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;
            
            if (src[ny * width + nx].w != 0) {
                dst[idx] = stroke_color;
                return;
            }
        }
    }
    dst[idx] = pixel;
}

__global__ void innerStrokeKernel(const uchar4* src,
                                  uchar4* dst,
                                  uint32_t width,
                                  uint32_t height,
                                  int stroke_width,
                                  uchar4 stroke_color) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 pixel = src[idx];

    if (pixel.w == 0) {
        dst[idx] = pixel;
        return;
    }
    
    int r2 = stroke_width * stroke_width;
    bool isBorder = false;
    
    if (x < stroke_width || x >= width - stroke_width || 
        y < stroke_width || y >= height - stroke_width) {
        isBorder = true;
    }
    
    if (!isBorder) {
        for (int dy = -stroke_width; dy <= stroke_width && !isBorder; dy++) {
            for (int dx = -stroke_width; dx <= stroke_width && !isBorder; dx++) {
                if (dx*dx + dy*dy > r2) continue;
                
                int nx = x + dx;
                int ny = y + dy;
                if (nx < 0 || nx >= width || ny < 0 || ny >= height) {
                    isBorder = true;
                } else if (src[ny * width + nx].w == 0) {
                    isBorder = true;
                }
            }
        }
    }
    
    dst[idx] = isBorder ? stroke_color : pixel;
}

__global__ void applyOpacityKernel(uchar4* buffer, 
                                   uint32_t width, 
                                   uint32_t height,
                                   float opacity) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    uchar4 pixel = buffer[idx];
    
    float currentAlpha = pixel.w / 255.0f;
    float newAlpha = currentAlpha * opacity;
    buffer[idx].w = static_cast<unsigned char>(newAlpha * 255.0f);
}

__global__ void flipKernel(uchar4* buffer,
                           uint32_t width,
                           uint32_t height,
                           bool flipHorizontal,
                           bool flipVertical) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    if ((flipHorizontal && x >= width/2) || 
        (flipVertical && y >= height/2)) return;
    
    int src_x = flipHorizontal ? (width - 1 - x) : x;
    int src_y = flipVertical ? (height - 1 - y) : y;
    
    if (src_x == x && src_y == y) return;
    
    int idx1 = y * width + x;
    int idx2 = src_y * width + src_x;
    
    uchar4 temp = buffer[idx1];
    buffer[idx1] = buffer[idx2];
    buffer[idx2] = temp;
}


__global__ void fillGradientKernel(uchar4* buffer, 
                                   uint32_t width,
                                   uint32_t height,
                                   unsigned char r1,
                                   unsigned char g1,
                                   unsigned char b1,
                                   unsigned char a1,
                                   unsigned char r2,
                                   unsigned char g2,
                                   unsigned char b2,
                                   unsigned char a2,
                                   int direction,
                                   bool seamless) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= (int)width || y >= (int)height) return;

    float factor = 0.0f;

    float nx = (float)x / (float)(width - 1) - 0.5f;
    float ny = (float)y / (float)(height - 1) - 0.5f;

    switch(direction) {
        case 0: // horizontal
            factor = (float)x / (float)(width - 1);
            break;
        case 1: // vertical
            factor = (float)y / (float)(height - 1);
            break;
        case 2: // diagonal
            float u = (float)x / (float)(width - 1);
            float v = (float)y / (float)(height - 1);
            factor = (u + v) * 0.5f;
            break;
        case 3: // radial
            factor = sqrtf(nx*nx + ny*ny) * 1.414f;
            factor = min(1.0f, factor);
            break;
    }

    if (seamless) {
        factor = factor < 0.5f ? 
                factor * 2.0f : 
                2.0f * (1.0f - factor);
    }

    unsigned char R = (unsigned char)(r1 + (r2 - r1) * factor);
    unsigned char G = (unsigned char)(g1 + (g2 - g1) * factor);
    unsigned char B = (unsigned char)(b1 + (b2 - b1) * factor);
    unsigned char A = (unsigned char)(a1 + (a2 - a1) * factor);

    int idx = y * width + x;
    buffer[idx] = make_uchar4(R, G, B, A);
}

extern "C" {

uchar4* create_buffer(uint32_t width,
                      uint32_t height) {
    uchar4* buffer;
    hipError_t err = hipMalloc(&buffer, width * height * sizeof(uchar4));
    if (err != hipSuccess) {
        printf("Error in hipMalloc: %s\n", hipGetErrorString(err));
        return nullptr;
    }
    hipDeviceSynchronize();
    return buffer;
}

void free_buffer(uchar4* buffer) {
    if (buffer) {
        hipFree(buffer);
    }
    hipDeviceSynchronize();
}

void copy_buffers_same_size(uchar4* dst,
                            const uchar4* src,
                            uint32_t width,
                            uint32_t height) {
    if (!dst || !src) {
        printf("Error: Null pointer provided to copy_buffers_same_size\n");
        return;
    }
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    
    copyBufferKernel<<<grid, block>>>(dst, src, width, height);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error in copy_buffers_same_size: %s\n", hipGetErrorString(err));
    }
    
    hipDeviceSynchronize();
}

void copy_to_device(uchar4* d_dst,
                    const uchar4* h_src,
                    uint32_t width,
                    uint32_t height) {
    if (!d_dst || !h_src) return;

    hipMemcpy(d_dst, h_src, width * height * sizeof(uchar4), 
               hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
}

void copy_to_host(uchar4* h_dst,
                  const uchar4* d_src,
                  uint32_t width,
                  uint32_t height) {
    if (!h_dst || !d_src) return;

    hipMemcpy(h_dst, d_src, width * height * sizeof(uchar4), 
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}

void blend_buffers(uchar4* dst,
                   const uchar4* src,
                   uint32_t dst_width,
                   uint32_t dst_height,
                   uint32_t src_width,
                   uint32_t src_height,
                   int32_t x,
                   int32_t y) {          
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);
              
    blendKernel<<<grid, block>>>(dst, src, dst_width, dst_height,
                                src_width, src_height, x, y);

    hipDeviceSynchronize();
}

void resize_bilinear(uchar4* dst,
                     const uchar4* src,
                     uint32_t dst_width,
                     uint32_t dst_height,
                     uint32_t src_width,
                     uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);
              
    resizeBilinearKernel<<<grid, block>>>(dst, src,
                                         dst_width, dst_height,
                                         src_width, src_height);
    
    hipDeviceSynchronize();
}

void resize_nearest(uchar4* dst,
                    const uchar4* src,
                    uint32_t dst_width,
                    uint32_t dst_height,
                    uint32_t src_width,
                    uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
                (dst_height + block.y - 1) / block.y);
                
    resizeNearestKernel<<<grid, block>>>(dst, src,
                                        dst_width, dst_height,
                                        src_width, src_height);
    
    hipDeviceSynchronize();
}

void resize_bicubic(uchar4* dst,
                    const uchar4* src,
                    uint32_t dst_width,
                    uint32_t dst_height,
                    uint32_t src_width,
                    uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
                (dst_height + block.y - 1) / block.y);
            
    resizeBicubicKernel<<<grid, block>>>(dst, src,
                                        dst_width, dst_height,
                                        src_width, src_height);

    hipDeviceSynchronize();
}

void fill_color(uchar4* buffer,
                uint32_t width,
                uint32_t height,
                unsigned char r,
                unsigned char g,
                unsigned char b,
                unsigned char a) {
    if (!buffer) return;

    uchar4 color = make_uchar4(r, g, b, a);
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
              
    fillColorKernel<<<grid, block>>>(buffer, color, width, height);

    hipDeviceSynchronize();
}

void apply_corner_radius(uchar4* buffer,
                         uint32_t width,
                         uint32_t height,
                         uint32_t size) {
    if (!buffer) return;

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y);
                
    cornerRadiusKernel<<<grid, block>>>(buffer, width, height, size);

    hipDeviceSynchronize();
}

void apply_stroke(uchar4* buffer,
                  const uchar4* copy_buffer,
                  uint32_t width,
                  uint32_t height,
                  int stroke_width,
                  unsigned char stroke_r,
                  unsigned char stroke_g,
                  unsigned char stroke_b,
                  unsigned char stroke_a,
                  int mode) {

    uchar4 stroke_color = make_uchar4(stroke_r, stroke_g, stroke_b, stroke_a);
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
    
    if (mode == 0) {
        strokeKernel<<<grid, block>>>(copy_buffer, buffer, width, height,
                                        stroke_width, stroke_color);
    } else if (mode == 1) {
        innerStrokeKernel<<<grid, block>>>(copy_buffer, buffer, width, height,
                                             stroke_width, stroke_color);
    }
    
    hipDeviceSynchronize();
}


void apply_opacity(uchar4* buffer,
                   uint32_t width,
                   uint32_t height,
                   float opacity) {
    if (!buffer) return;
    
    opacity = min(max(opacity, 0.0f), 1.0f);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);
            
    applyOpacityKernel<<<grid, block>>>(buffer, width, height, opacity);
    hipDeviceSynchronize();
}

void apply_shadow(uchar4* buffer,
                  const uchar4* copy_buffer,
                  uint32_t width,
                  uint32_t height,
                  float radius,
                  float intensity,
                  unsigned char shadow_r,
                  unsigned char shadow_g,
                  unsigned char shadow_b,
                  unsigned char shadow_a,
                  int mode) {
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
    
    uchar4 shadow_color = make_uchar4(shadow_r, shadow_g, shadow_b, shadow_a);
    bool isInner = mode == 1;
    
    shadowKernel<<<grid, block>>>(copy_buffer, buffer,
                                  width, height,
                                  radius, intensity,
                                  shadow_color, isInner);
    
    hipDeviceSynchronize();
}


void apply_flip(uchar4* buffer,
                uint32_t width,
                uint32_t height,
                bool flip_horizontal,
                bool flip_vertical) {
    if (!buffer) return;
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
              
    flipKernel<<<grid, block>>>(buffer, width, height,
                               flip_horizontal, flip_vertical);
    
    hipDeviceSynchronize();
}

void apply_grayscale(uchar4* buffer,
                    uint32_t width,
                    uint32_t height) {
    if (!buffer) return;
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
              
    grayscaleKernel<<<grid, block>>>(buffer, width, height);
    
    hipDeviceSynchronize();
}

void crop_image(uchar4* dst,
                const uchar4* src,
                uint32_t src_width,
                uint32_t src_height,
                uint32_t dst_width,
                uint32_t dst_height,
                int crop_x,
                int crop_y) {
    if (!src || !dst) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);

    cropKernel<<<grid, block>>>(src, dst,
                                src_width, src_height,
                                dst_width, dst_height,
                                crop_x, crop_y);
    hipDeviceSynchronize();
}

void fill_gradient(uchar4* buffer,
                   uint32_t width,
                   uint32_t height,
                   unsigned char r1,
                   unsigned char g1,
                   unsigned char b1,
                   unsigned char a1,
                   unsigned char r2,
                   unsigned char g2,
                   unsigned char b2,
                   unsigned char a2,
                   int direction,
                   bool seamless) {
    if (!buffer) return;

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    fillGradientKernel<<<grid, block>>>(buffer, width, height,
                                            r1, g1, b1, a1,
                                            r2, g2, b2, a2,
                                            direction, seamless);
    hipDeviceSynchronize();
}

void apply_gaussian_blur(uchar4* buffer,
                         const uchar4* copy_buffer,
                         uint32_t width,
                         uint32_t height,
                         float radius) {
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
    
    gaussianBlurKernel<<<grid, block>>>(copy_buffer, buffer, width, height, radius);
    
    hipDeviceSynchronize();
}

void apply_chroma_key(uchar4* buffer,
                    const uchar4* key_buffer,
                    uint32_t buffer_width,
                    uint32_t buffer_height,
                    uint32_t key_width,
                    uint32_t key_height,
                    int channel,
                    unsigned char threshold,
                    bool invert,
                    bool zero_all_channels) {
    if (!buffer || !key_buffer) return;
    
    dim3 block(16, 16);
    dim3 grid((buffer_width + block.x - 1) / block.x,
              (buffer_height + block.y - 1) / block.y);
              
    chromaKeyKernel<<<grid, block>>>(buffer, key_buffer, 
                                     buffer_width, buffer_height, 
                                     key_width, key_height, 
                                     channel, threshold, invert,
                                     zero_all_channels);
    
    hipDeviceSynchronize();
}

}