#include "hip/hip_runtime.h"
#include "photoff.h"
#include <stdio.h>


__global__ void fillColorKernel(uchar4* buffer,
                                uchar4 color, 
                                uint32_t width,
                                uint32_t height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        buffer[idx] = color;
    }
}


__global__ void blendKernel(uchar4* dst,
                            const uchar4* src,
                            uint32_t dst_width,
                            uint32_t dst_height,
                            uint32_t src_width,
                            uint32_t src_height,
                            int32_t pos_x,
                            int32_t pos_y) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_width || y >= dst_height) return;

    int src_x = x - pos_x;
    int src_y = y - pos_y;

    if (src_x >= 0 && src_x < src_width && 
        src_y >= 0 && src_y < src_height) {
        
        int dst_idx = y * dst_width + x;
        int src_idx = src_y * src_width + src_x;
        
        uchar4 src_pixel = src[src_idx];
        uchar4 dst_pixel = dst[dst_idx];
        
        float srcA = src_pixel.w / 255.0f;
        float dstA = dst_pixel.w / 255.0f;

        // Case 1: pixel totally opaque
        if (src_pixel.w == 255) {
            dst[dst_idx] = src_pixel;
        } 
        // Case 2: pixel totally transparent
        else if (src_pixel.w == 0) {
            // Do nothing
        } 
        // Case 3: pixel semi-transparent
        else {
            float outA = srcA + dstA * (1.0f - srcA);

            // Avoid division by zero
            if (outA > 0.0f) {
                float outR = (src_pixel.x * srcA + dst_pixel.x * dstA * (1.0f - srcA)) / outA;
                float outG = (src_pixel.y * srcA + dst_pixel.y * dstA * (1.0f - srcA)) / outA;
                float outB = (src_pixel.z * srcA + dst_pixel.z * dstA * (1.0f - srcA)) / outA;

                dst[dst_idx].x = static_cast<unsigned char>(outR);
                dst[dst_idx].y = static_cast<unsigned char>(outG);
                dst[dst_idx].z = static_cast<unsigned char>(outB);
                dst[dst_idx].w = static_cast<unsigned char>(outA * 255.0f);
            } else {
                // Do nothing
            }
        }
    }
}

__global__ void cornerRadiusKernel(uchar4* buffer,
                                   uint32_t width,
                                   uint32_t height,
                                   uint32_t radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    
    if (x < radius && y < radius) {
        int dx = radius - 1 - x;
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y < radius) {
        int dx = x - (width - radius);
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x < radius && y >= height - radius) {
        int dx = radius - 1 - x;
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y >= height - radius) {
        int dx = x - (width - radius);
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }
}


extern "C" {

uchar4* create_buffer(uint32_t width,
                      uint32_t height) {

    uchar4* buffer;
    hipError_t err = hipMalloc(&buffer, width * height * sizeof(uchar4));
    if (err != hipSuccess) {
        printf("Error in hipMalloc: %s\n", hipGetErrorString(err));
        return nullptr;
    }
    hipDeviceSynchronize();
    return buffer;
}

void free_buffer(uchar4* buffer) {
    if (buffer) {
        hipFree(buffer);
    }
    hipDeviceSynchronize();
}

void copy_to_device(uchar4* d_dst,
                    const uchar4* h_src,
                    uint32_t width,
                    uint32_t height) {

    if (!d_dst || !h_src) return;

    hipMemcpy(d_dst, h_src, width * height * sizeof(uchar4), 
               hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
}

void copy_to_host(uchar4* h_dst,
                  const uchar4* d_src,
                  uint32_t width,
                  uint32_t height) {

    if (!h_dst || !d_src) return;

    hipMemcpy(h_dst, d_src, width * height * sizeof(uchar4), 
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}

void blend_buffers(uchar4* dst,
                   const uchar4* src,
                   uint32_t dst_width,
                   uint32_t dst_height,
                   uint32_t src_width,
                   uint32_t src_height,
                   int32_t x,
                   int32_t y) {
                    
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);
              
    blendKernel<<<grid, block>>>(dst, src, dst_width, dst_height,
                                src_width, src_height, x, y);

    hipDeviceSynchronize();
}

void fill_color(uchar4* buffer,
                uint32_t width,
                uint32_t height,
                unsigned char r,
                unsigned char g,
                unsigned char b,
                unsigned char a) {

    if (!buffer) return;

    uchar4 color = make_uchar4(r, g, b, a);
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
              
    fillColorKernel<<<grid, block>>>(buffer, color, width, height);

    hipDeviceSynchronize();
}

void apply_corner_radius(uchar4* buffer,
                         uint32_t width,
                         uint32_t height,
                         uint32_t size) {
    if (!buffer) return;

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y);
                
    cornerRadiusKernel<<<grid, block>>>(buffer, width, height, size);

    hipDeviceSynchronize();
}

}