#include "hip/hip_runtime.h"
#include "photoff.h"
#include <stdio.h>

__device__ float bicubicWeight(float x, float a = -0.5f) {
    x = fabsf(x);
    if (x <= 1.0f) {
        return ((a + 2.0f) * x * x * x) - ((a + 3.0f) * x * x) + 1.0f;
    } else if (x < 2.0f) {
        return (a * x * x * x) - (5.0f * a * x * x) + (8.0f * a * x) - (4.0f * a);
    }
    return 0.0f;
}

__global__ void resizeBicubicKernel(uchar4* dst,
                                   const uchar4* src,
                                   uint32_t dst_width,
                                   uint32_t dst_height,
                                   uint32_t src_width,
                                   uint32_t src_height) {
    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)(src_width) / dst_width;
    float scale_y = (float)(src_height) / dst_height;
    
    float src_x = dst_x * scale_x;
    float src_y = dst_y * scale_y;

    int x0 = floorf(src_x - 1.0f);
    int y0 = floorf(src_y - 1.0f);
    
    float4 result = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    float totalWeight = 0.0f;

    #pragma unroll
    for (int dy = 0; dy < 4; dy++) {
        int sy = y0 + dy;
        float wy = bicubicWeight(src_y - sy);
        
        #pragma unroll
        for (int dx = 0; dx < 4; dx++) {
            int sx = x0 + dx;
            
            if (sx >= 0 && sx < src_width && sy >= 0 && sy < src_height) {
                float wx = bicubicWeight(src_x - sx);
                float weight = wx * wy;
                
                uchar4 pixel = src[sy * src_width + sx];
                result.x += weight * pixel.x;
                result.y += weight * pixel.y;
                result.z += weight * pixel.z;
                result.w += weight * pixel.w;
                totalWeight += weight;
            }
        }
    }

    if (totalWeight > 0.0f) {
        result.x = fmaxf(0.0f, fminf(255.0f, result.x / totalWeight));
        result.y = fmaxf(0.0f, fminf(255.0f, result.y / totalWeight));
        result.z = fmaxf(0.0f, fminf(255.0f, result.z / totalWeight));
        result.w = fmaxf(0.0f, fminf(255.0f, result.w / totalWeight));
    }

    dst[dst_y * dst_width + dst_x] = make_uchar4(
        __float2int_rn(result.x),
        __float2int_rn(result.y),
        __float2int_rn(result.z),
        __float2int_rn(result.w)
    );
}

__global__ void resizeBilinearKernel(uchar4* dst,
                                     const uchar4* src,
                                     uint32_t dst_width,
                                     uint32_t dst_height,
                                     uint32_t src_width,
                                     uint32_t src_height) {

    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)(src_width - 1) / dst_width;
    float scale_y = (float)(src_height - 1) / dst_height;
    
    float src_x = dst_x * scale_x;
    float src_y = dst_y * scale_y;
    
    int x1 = (int)src_x;
    int y1 = (int)src_y;
    int x2 = min(x1 + 1, (int)src_width - 1);
    int y2 = min(y1 + 1, (int)src_height - 1);
    
    float wx2 = src_x - x1;
    float wy2 = src_y - y1;
    float wx1 = 1.0f - wx2;
    float wy1 = 1.0f - wy2;
    
    uchar4 p11 = src[y1 * src_width + x1];
    uchar4 p21 = src[y1 * src_width + x2];
    uchar4 p12 = src[y2 * src_width + x1];
    uchar4 p22 = src[y2 * src_width + x2];
    
    int dst_idx = dst_y * dst_width + dst_x;
    dst[dst_idx].x = (unsigned char)(
        p11.x * wx1 * wy1 +
        p21.x * wx2 * wy1 +
        p12.x * wx1 * wy2 +
        p22.x * wx2 * wy2);
    
    dst[dst_idx].y = (unsigned char)(
        p11.y * wx1 * wy1 +
        p21.y * wx2 * wy1 +
        p12.y * wx1 * wy2 +
        p22.y * wx2 * wy2);
    
    dst[dst_idx].z = (unsigned char)(
        p11.z * wx1 * wy1 +
        p21.z * wx2 * wy1 +
        p12.z * wx1 * wy2 +
        p22.z * wx2 * wy2);
    
    dst[dst_idx].w = (unsigned char)(
        p11.w * wx1 * wy1 +
        p21.w * wx2 * wy1 +
        p12.w * wx1 * wy2 +
        p22.w * wx2 * wy2);
}

__global__ void resizeNearestKernel(uchar4* dst,
                                    const uchar4* src,
                                    uint32_t dst_width,
                                    uint32_t dst_height,
                                    uint32_t src_width,
                                    uint32_t src_height) {

    int dst_x = blockIdx.x * blockDim.x + threadIdx.x;
    int dst_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (dst_x >= dst_width || dst_y >= dst_height) return;

    float scale_x = (float)src_width / dst_width;
    float scale_y = (float)src_height / dst_height;

    int src_x = (int)(dst_x * scale_x);
    int src_y = (int)(dst_y * scale_y);
    
    dst[dst_y * dst_width + dst_x] = src[src_y * src_width + src_x];
}


__global__ void fillColorKernel(uchar4* buffer,
                                uchar4 color, 
                                uint32_t width,
                                uint32_t height) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;
        buffer[idx] = color;
    }
}


__global__ void blendKernel(uchar4* dst,
                            const uchar4* src,
                            uint32_t dst_width,
                            uint32_t dst_height,
                            uint32_t src_width,
                            uint32_t src_height,
                            int32_t pos_x,
                            int32_t pos_y) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_width || y >= dst_height) return;

    int src_x = x - pos_x;
    int src_y = y - pos_y;

    if (src_x >= 0 && src_x < src_width && 
        src_y >= 0 && src_y < src_height) {
        
        int dst_idx = y * dst_width + x;
        int src_idx = src_y * src_width + src_x;
        
        uchar4 src_pixel = src[src_idx];
        uchar4 dst_pixel = dst[dst_idx];
        
        float srcA = src_pixel.w / 255.0f;
        float dstA = dst_pixel.w / 255.0f;

        // Case 1: pixel totally opaque
        if (src_pixel.w == 255) {
            dst[dst_idx] = src_pixel;
        } 
        // Case 2: pixel totally transparent
        else if (src_pixel.w == 0) {
            // Do nothing
        } 
        // Case 3: pixel semi-transparent
        else {
            float outA = srcA + dstA * (1.0f - srcA);

            // Avoid division by zero
            if (outA > 0.0f) {
                float outR = (src_pixel.x * srcA + dst_pixel.x * dstA * (1.0f - srcA)) / outA;
                float outG = (src_pixel.y * srcA + dst_pixel.y * dstA * (1.0f - srcA)) / outA;
                float outB = (src_pixel.z * srcA + dst_pixel.z * dstA * (1.0f - srcA)) / outA;

                dst[dst_idx].x = static_cast<unsigned char>(outR);
                dst[dst_idx].y = static_cast<unsigned char>(outG);
                dst[dst_idx].z = static_cast<unsigned char>(outB);
                dst[dst_idx].w = static_cast<unsigned char>(outA * 255.0f);
            } else {
                // Do nothing
            }
        }
    }
}

__global__ void cornerRadiusKernel(uchar4* buffer,
                                   uint32_t width,
                                   uint32_t height,
                                   uint32_t radius) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    
    if (x < radius && y < radius) {
        int dx = radius - 1 - x;
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y < radius) {
        int dx = x - (width - radius);
        int dy = radius - 1 - y;
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x < radius && y >= height - radius) {
        int dx = radius - 1 - x;
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }

    else if (x >= width - radius && y >= height - radius) {
        int dx = x - (width - radius);
        int dy = y - (height - radius);
        if (dx * dx + dy * dy > radius * radius) {
            buffer[idx] = make_uchar4(0, 0, 0, 0);
        }
    }
}

__global__ void strokeKernel(const uchar4* src,
                             uchar4* dst,
                             uint32_t width,
                             uint32_t height,
                             int stroke_width,
                             uchar4 stroke_color) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 pixel = src[idx];

    if (pixel.w != 0) {
        dst[idx] = pixel;
        return;
    }
    
    int r2 = stroke_width * stroke_width;
    for (int dy = -stroke_width; dy <= stroke_width; dy++) {
        for (int dx = -stroke_width; dx <= stroke_width; dx++) {
            if (dx*dx + dy*dy > r2) continue;
            
            int nx = x + dx;
            int ny = y + dy;
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;
            
            if (src[ny * width + nx].w != 0) {
                dst[idx] = stroke_color;
                return;
            }
        }
    }
    dst[idx] = pixel;
}

__global__ void innerStrokeKernel(const uchar4* src,
                                  uchar4* dst,
                                  uint32_t width,
                                  uint32_t height,
                                  int stroke_width,
                                  uchar4 stroke_color) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    uchar4 pixel = src[idx];

    if (pixel.w == 0) {
        dst[idx] = pixel;
        return;
    }
    
    int r2 = stroke_width * stroke_width;
    bool isBorder = false;
    
    for (int dy = -stroke_width; dy <= stroke_width && !isBorder; dy++) {
        for (int dx = -stroke_width; dx <= stroke_width && !isBorder; dx++) {
            if (dx*dx + dy*dy > r2) continue;
            
            int nx = x + dx;
            int ny = y + dy;
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;
            
            if (src[ny * width + nx].w == 0) {
                isBorder = true;
            }
        }
    }
    
    dst[idx] = isBorder ? stroke_color : pixel;
}


__global__ void applyOpacityKernel(uchar4* buffer, 
                                  uint32_t width, 
                                  uint32_t height,
                                  float opacity) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    uchar4 pixel = buffer[idx];
    
    float currentAlpha = pixel.w / 255.0f;
    float newAlpha = currentAlpha * opacity;
    buffer[idx].w = static_cast<unsigned char>(newAlpha * 255.0f);
}

extern "C" {

uchar4* create_buffer(uint32_t width,
                      uint32_t height) {

    uchar4* buffer;
    hipError_t err = hipMalloc(&buffer, width * height * sizeof(uchar4));
    if (err != hipSuccess) {
        printf("Error in hipMalloc: %s\n", hipGetErrorString(err));
        return nullptr;
    }
    hipDeviceSynchronize();
    return buffer;
}

void free_buffer(uchar4* buffer) {
    if (buffer) {
        hipFree(buffer);
    }
    hipDeviceSynchronize();
}

void copy_to_device(uchar4* d_dst,
                    const uchar4* h_src,
                    uint32_t width,
                    uint32_t height) {

    if (!d_dst || !h_src) return;

    hipMemcpy(d_dst, h_src, width * height * sizeof(uchar4), 
               hipMemcpyHostToDevice);
    
    hipDeviceSynchronize();
}

void copy_to_host(uchar4* h_dst,
                  const uchar4* d_src,
                  uint32_t width,
                  uint32_t height) {

    if (!h_dst || !d_src) return;

    hipMemcpy(h_dst, d_src, width * height * sizeof(uchar4), 
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}

void blend_buffers(uchar4* dst,
                   const uchar4* src,
                   uint32_t dst_width,
                   uint32_t dst_height,
                   uint32_t src_width,
                   uint32_t src_height,
                   int32_t x,
                   int32_t y) {
                    
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);
              
    blendKernel<<<grid, block>>>(dst, src, dst_width, dst_height,
                                src_width, src_height, x, y);

    hipDeviceSynchronize();
}

void resize_bilinear(uchar4* dst,
                     const uchar4* src,
                     uint32_t dst_width,
                     uint32_t dst_height,
                     uint32_t src_width,
                     uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
              (dst_height + block.y - 1) / block.y);
              
    resizeBilinearKernel<<<grid, block>>>(dst, src,
                                         dst_width, dst_height,
                                         src_width, src_height);
    
    hipDeviceSynchronize();
}

void resize_nearest(uchar4* dst,
                    const uchar4* src,
                    uint32_t dst_width,
                    uint32_t dst_height,
                    uint32_t src_width,
                    uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
                (dst_height + block.y - 1) / block.y);
                
    resizeNearestKernel<<<grid, block>>>(dst, src,
                                        dst_width, dst_height,
                                        src_width, src_height);
    
    hipDeviceSynchronize();
}

void resize_bicubic(uchar4* dst,
                    const uchar4* src,
                    uint32_t dst_width,
                    uint32_t dst_height,
                    uint32_t src_width,
                    uint32_t src_height) {
    if (!dst || !src) return;

    dim3 block(16, 16);
    dim3 grid((dst_width + block.x - 1) / block.x,
                (dst_height + block.y - 1) / block.y);
            
    resizeBicubicKernel<<<grid, block>>>(dst, src,
                                        dst_width, dst_height,
                                        src_width, src_height);

    hipDeviceSynchronize();
}

void fill_color(uchar4* buffer,
                uint32_t width,
                uint32_t height,
                unsigned char r,
                unsigned char g,
                unsigned char b,
                unsigned char a) {

    if (!buffer) return;

    uchar4 color = make_uchar4(r, g, b, a);
    
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);
              
    fillColorKernel<<<grid, block>>>(buffer, color, width, height);

    hipDeviceSynchronize();
}

void apply_corner_radius(uchar4* buffer,
                         uint32_t width,
                         uint32_t height,
                         uint32_t size) {
    if (!buffer) return;

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y);
                
    cornerRadiusKernel<<<grid, block>>>(buffer, width, height, size);

    hipDeviceSynchronize();
}

void apply_stroke(uchar4* buffer,
                  uint32_t width,
                  uint32_t height,
                  int stroke_width,
                  unsigned char stroke_r,
                  unsigned char stroke_g,
                  unsigned char stroke_b,
                  unsigned char stroke_a,
                  int mode) {
        if (!buffer) return;
        
        uchar4 stroke_color = make_uchar4(stroke_r, stroke_g, stroke_b, stroke_a);
        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x,
                  (height + block.y - 1) / block.y);
        
        uchar4* temp_buffer = nullptr;
        hipMalloc(&temp_buffer, width * height * sizeof(uchar4));
        if (!temp_buffer) return;
        
        hipMemcpy(temp_buffer, buffer, width * height * sizeof(uchar4),
                   hipMemcpyDeviceToDevice);
        
        if (mode == 0) {
            strokeKernel<<<grid, block>>>(temp_buffer, buffer, width, height,
                                          stroke_width, stroke_color);
        } else if (mode == 1) {
            innerStrokeKernel<<<grid, block>>>(temp_buffer, buffer, width, height,
                                               stroke_width, stroke_color);
        }
        
        hipDeviceSynchronize();
        hipFree(temp_buffer);
    }

    void apply_opacity(uchar4* buffer,
                       uint32_t width,
                       uint32_t height,
                       float opacity) {
        if (!buffer) return;
        
        opacity = min(max(opacity, 0.0f), 1.0f);

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y);
                
        applyOpacityKernel<<<grid, block>>>(buffer, width, height, opacity);
        hipDeviceSynchronize();
    }

}